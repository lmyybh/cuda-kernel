#include <hipblas.h>
#include "cmdline.h"
#include "helper_cuda.cuh"
#include "helper_data.h"

int main(int argc, char* argv[]) {
  cmdline::parser args;
  args.add<int>("height", 'h', "heigth of matrix", false, 32 * 300);
  args.add<int>("width", 'w', "width of matrix", false, 32 * 300);
  args.add<int>("device", 'd', "gpu id", false, 0);
  args.parse_check(argc, argv);

  const int M = args.get<int>("height");
  const int N = args.get<int>("width");
  const int nBytes = M * N * sizeof(float);

  float* A = nullptr;
  checkCudaErrors(hipHostMalloc((void**)&A, nBytes));
  initialRangeData(A, M * N, 0.0f, 1.0f);

  float* A_colmajor = nullptr;
  checkCudaErrors(hipHostMalloc((void**)&A_colmajor, nBytes));

  // 转换为列主序
  for (int col = 0; col < N; ++col)
    for (int row = 0; row < M; ++row) { A_colmajor[col * M + row] = A[row * N + col]; }

  float* d_A = nullptr;
  checkCudaErrors(hipMalloc((void**)&d_A, nBytes));
  float* d_B = nullptr;
  checkCudaErrors(hipMalloc((void**)&d_B, nBytes));
  checkCudaErrors(hipMemcpy(d_A, A_colmajor, nBytes, hipMemcpyHostToDevice));

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float alpha = 1.0f, beta = 0.0f;

  hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, &alpha, d_A, M, &beta, nullptr, N, d_B, N);

  hipDeviceSynchronize();

  float* gpuB = nullptr;
  checkCudaErrors(hipHostMalloc((void**)&gpuB, nBytes));
  checkCudaErrors(hipMemcpy(gpuB, d_B, nBytes, hipMemcpyDeviceToHost));

  //   print2D<float>(A_colmajor, N, M);
  //   printf("-------------------------\n");
  //   print2D<float>(gpuB, M, N);

  checkCudaErrors(hipHostFree(A));
  checkCudaErrors(hipHostFree(A_colmajor));
  checkCudaErrors(hipHostFree(gpuB));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));

  return 0;
}