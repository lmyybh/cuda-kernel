#include <hipblas.h>
#include "cmdline.h"
#include "helper_cuda.cuh"
#include "helper_data.h"

int main(int argc, char* argv[]) {
  cmdline::parser args;
  args.add<int>("M", 'm', "rows of matrix A", false, 2048);
  args.add<int>("K", 'k', "columns of matrix A", false, 2048);
  args.add<int>("N", 'n', "columns of matrix B", false, 2048);
  args.add<int>("device", 'd', "gpu id", false, 0);
  args.parse_check(argc, argv);

  // 矩阵尺寸
  const int M = args.get<int>("M");
  const int K = args.get<int>("K");
  const int N = args.get<int>("N");
  printf("M: [%d], K: [%d], N: [%d]\n", M, K, N);

  float alpha = 1.0f;
  float beta = 0.0f;

  const int A_Bytes = M * K * sizeof(float);
  const int B_Bytes = K * N * sizeof(float);
  const int out_Bytes = M * N * sizeof(float);

  // 初始化 host 数据
  float* A = nullptr;
  checkCudaErrors(hipHostMalloc((void**)&A, A_Bytes));
  normalInitialData(A, M * K, 1);

  float* B = nullptr;
  checkCudaErrors(hipHostMalloc((void**)&B, B_Bytes));
  normalInitialData(B, K * N, 2);

  float* out = nullptr;
  checkCudaErrors(hipHostMalloc((void**)&out, out_Bytes));
  initialRangeData(out, M * N, 0, 0);

  // 数据拷贝: host -> device
  float* d_A = nullptr;
  checkCudaErrors(hipMalloc((void**)&d_A, A_Bytes));
  checkCudaErrors(hipMemcpy(d_A, A, A_Bytes, hipMemcpyHostToDevice));

  float* d_B = nullptr;
  checkCudaErrors(hipMalloc((void**)&d_B, B_Bytes));
  checkCudaErrors(hipMemcpy(d_B, B, B_Bytes, hipMemcpyHostToDevice));

  float* d_out = nullptr;
  checkCudaErrors(hipMalloc((void**)&d_out, out_Bytes));
  checkCudaErrors(hipMemcpy(d_out, out, out_Bytes, hipMemcpyHostToDevice));

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_out, N);

  float* gpuOut = nullptr;
  checkCudaErrors(hipHostMalloc((void**)&gpuOut, out_Bytes));
  checkCudaErrors(hipMemcpy(gpuOut, d_out, out_Bytes, hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  // 释放资源
  checkCudaErrors(hipHostFree(A));
  checkCudaErrors(hipHostFree(B));
  checkCudaErrors(hipHostFree(out));
  checkCudaErrors(hipHostFree(gpuOut));

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_out));

  return 0;
}
