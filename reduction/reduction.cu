
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WARP_SIZE 32
#define CEIL(a, b) (((a) + (b) - 1) / (b))
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// version 0: 简单实现，存在线程束分化问题
__global__ void reduce0(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 读取数据到 shared memory
  int tid = threadIdx.x;
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  data[tid] = index < N ? d_A[index] : 0.f;

  // 同步，保证读取完成
  __syncthreads();

  // iter0 [s = 1] (t0, t2, t4, ...): t0 -> (0, 1) | t2 -> (2, 3) | t4 -> (4, 5) ...
  // iter1 [s = 2] (t0, t4, t8, ...): t0 -> (0, 2) | t4 -> (4, 6) | t8 -> (8, 10) ...
  for (int s = 1; s < blockDim.x; s <<= 1) {
    // 负责执行运算的 threadIdx 为 0, 2s, 4s, 8s, ...
    if ((tid % (s * 2)) == 0) { data[tid] += data[tid + s]; }

    // 进行同步，保证计算完成
    __syncthreads();
  }

  // block 负责数据的求和结果存储在 data[0]，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = data[0]; }
}

// version 0.5: 优化取余运算
__global__ void reduce0_5(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 读取数据到 shared memory
  int tid = threadIdx.x;
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  data[tid] = index < N ? d_A[index] : 0.f;

  // 同步，保证读取完成
  __syncthreads();

  // iter0 [s = 1] (t0, t2, t4, ...): t0 -> (0, 1) | t2 -> (2, 3) | t4 -> (4, 5) ...
  // iter1 [s = 2] (t0, t4, t8, ...): t0 -> (0, 2) | t4 -> (4, 6) | t8 -> (8, 10) ...
  for (int s = 1; s < blockDim.x; s <<= 1) {
    // 负责执行运算的 threadIdx 为 0, 2s, 4s, 8s, ...
    if ((tid & (s * 2 - 1)) == 0) { data[tid] += data[tid + s]; }

    // 进行同步，保证计算完成
    __syncthreads();
  }

  // block 负责数据的求和结果存储在 data[0]，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = data[0]; }
}

// version1: 使用连续 thread 负责计算，解决线程束分化，存在 bank conflicts
__global__ void reduce1(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 读取数据到 shared memory
  int tid = threadIdx.x;
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  data[tid] = i < N ? d_A[i] : 0.f;

  // 同步，保证读取完成
  __syncthreads();

  // iter0 [s = 1] (t0 ~ tN/2): t0 -> (0, 1) | t1 -> (2, 3) | t2 -> (4, 5) ...
  // iter1 [s = 2] (t0 ~ tN/4): t0 -> (0, 2) | t1 -> (4, 6) | t2 -> (8, 10) ...
  for (int s = 1; s < blockDim.x; s <<= 1) {
    // 使用连续 thread 执行运算，需要设置 blockDim.x 为偶数，避免越界
    int index = tid * s * 2;
    if (index < blockDim.x) { data[index] += data[index + s]; }

    // 进行同步，保证计算完成
    __syncthreads();
  }

  // block 负责数据的求和结果存储在 data[0]，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = data[0]; }
}

// version2: 步长从大到小变化，解决 bank conflicts
__global__ void reduce2(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 读取数据到 shared memory
  int tid = threadIdx.x;
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  data[tid] = i < N ? d_A[i] : 0.f;

  // 同步，保证读取完成
  __syncthreads();

  // iter0 [s = N/2] (t0 ~ tN/2-1): t0 -> (0, N/2) | t1 -> (1, N/2 + 1) | t2 -> (2, N/2 + 2) ...
  // iter1 [s = N/4] (t0 ~ tN/4-1): t0 -> (0, N/4) | t1 -> (1, N/4 + 1) | t2 -> (2, N/4 + 2) ...
  // stride 从大到小变化，避免 bank conflicts
  for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
    if (tid < s) { data[tid] += data[tid + s]; }

    // 进行同步，保证计算完成
    __syncthreads();
  }

  // block 负责数据的求和结果存储在 data[0]，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = data[0]; }
}

// version3: 读取数据到 shared memory 时，进行一次加法计算
__global__ void reduce3(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 计算索引
  int tid = threadIdx.x;
  int i = 2 * blockDim.x * blockIdx.x + threadIdx.x;  // 每个 block 负责 2 * blockDim.x 个元素

  // 将第 i 和 i + blockDim.x 个元素求和，结果写入到 shared memory 中（需要避免越界）
  float sum = i < N ? d_A[i] : 0.f;
  if (i + blockDim.x < N) { sum += d_A[i + blockDim.x]; }
  data[tid] = sum;

  // 同步，保证读取完成
  __syncthreads();

  // stride 从大到小变化，避免 bank conflicts
  for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
    if (tid < s) { data[tid] += data[tid + s]; }

    // 进行同步，保证计算完成
    __syncthreads();
  }

  // block 负责数据的求和结果存储在 data[0]，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = data[0]; }
}

// 使用 __shfl_xor_sync 实现 WarpReduce
template<int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warpReduce(float val) {
#pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// version4: 最后 32 个数使用 warp shuffle 完成求和
__global__ void reduce4(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 计算索引
  int tid = threadIdx.x;
  int i = 2 * blockDim.x * blockIdx.x + threadIdx.x;  // 每个 block 负责 2 * blockDim.x 个元素

  // 将第 i 和 i + blockDim.x 个元素求和，结果写入到 shared memory 中（需要避免越界）
  float sum = i < N ? d_A[i] : 0.f;
  if (i + blockDim.x < N) { sum += d_A[i + blockDim.x]; }
  data[tid] = sum;

  // 同步，保证读取完成
  __syncthreads();

  // stride 从大到小变化，避免 bank conflicts，循环条件变为 s >= 32
  for (int s = blockDim.x >> 1; s >= 32; s >>= 1) {
    if (tid < s) { data[tid] = sum = sum + data[tid + s]; }

    // 进行同步，保证计算完成
    __syncthreads();
  }

  // WarpReduce
  if (tid < 32) { sum = warpReduce<WARP_SIZE>(sum); }

  // block 负责数据的求和结果为 sum，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = sum; }
}

// version5: 展开循环
template<int blockSize>  // 将 blockSize 作为模板参数，可以在编译期确定其数值，进而优化 if 分支
__global__ void reduce5(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 计算索引
  int tid = threadIdx.x;
  int i = 2 * blockSize * blockIdx.x + threadIdx.x;  // 每个 block 负责 2 * blockDim.x 个元素

  // 将第 i 和 i + blockDim.x 个元素求和，结果写入到 shared memory 中（需要避免越界）
  float sum = i < N ? d_A[i] : 0;
  if (i + blockSize < N) sum += d_A[i + blockSize];
  data[tid] = sum;

  // 同步，保证读取完成
  __syncthreads();

  // 依据 blockSize 大小，展开循环
  if (blockSize >= 1024 && tid < 512) { data[tid] = sum = sum + data[tid + 512]; }
  __syncthreads();

  if (blockSize >= 512 && tid < 256) { data[tid] = sum = sum + data[tid + 256]; }
  __syncthreads();

  if (blockSize >= 256 && tid < 128) { data[tid] = sum = sum + data[tid + 128]; }
  __syncthreads();

  if (blockSize >= 128 && tid < 64) { data[tid] = sum = sum + data[tid + 64]; }
  __syncthreads();

  if (blockSize >= 64 && tid < 32) { data[tid] = sum = sum + data[tid + 32]; }
  __syncthreads();

  // WarpReduce
  if (tid < 32) { sum = warpReduce<WARP_SIZE>(sum); }

  // block 负责数据的求和结果为 sum，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = sum; }
}

// version6:读取数据到 shared memory 时，进行多次加法运算
template<int blockSize>  // 将 blockSize 作为模板参数，可以在编译期确定其数值，进而优化 if 分支
__global__ void reduce6(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 计算索引
  int tid = threadIdx.x;
  int i = blockSize * blockIdx.x + threadIdx.x;

  // 网格跨步循环，对多个元素求和
  float sum = 0.f;
  for (int index = i; index < N; index += blockSize * gridDim.x) { sum += d_A[index]; }
  data[tid] = sum;

  // 同步，保证读取完成
  __syncthreads();

  // 依据 blockSize 大小，展开循环
  if (blockSize >= 1024 && tid < 512) { data[tid] = sum = sum + data[tid + 512]; }
  __syncthreads();

  if (blockSize >= 512 && tid < 256) { data[tid] = sum = sum + data[tid + 256]; }
  __syncthreads();

  if (blockSize >= 256 && tid < 128) { data[tid] = sum = sum + data[tid + 128]; }
  __syncthreads();

  if (blockSize >= 128 && tid < 64) { data[tid] = sum = sum + data[tid + 64]; }
  __syncthreads();

  if (blockSize >= 64 && tid < 32) { data[tid] = sum = sum + data[tid + 32]; }
  __syncthreads();

  // WarpReduce
  if (tid < 32) { sum = warpReduce<WARP_SIZE>(sum); }

  // block 负责数据的求和结果为 sum，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = sum; }
}

// version6_vec4: 使用 float4 读取数据并求和，然后写入到 shared memory 中
template<int blockSize>  // 将 blockSize 作为模板参数，可以在编译期确定其数值，进而优化 if 分支
__global__ void reduce6_vec4(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 计算索引
  int tid = threadIdx.x;
  int i = 4 * (blockSize * blockIdx.x + threadIdx.x);  // 注意索引要乘以 4

  // 向量化访存
  float sum = 0.f;
  if (i < N - 4) {
    float4 reg = FLOAT4(d_A[i]);
    sum = reg.x + reg.y + reg.z + reg.w;
  } else {  // 不足 4 个元素时进行特殊处理
    for (int j = i; j < N; ++j) { sum += d_A[j]; }
  }
  data[tid] = sum;

  // 同步，保证读取完成
  __syncthreads();

  // 依据 blockSize 大小，展开循环
  if (blockSize >= 1024 && tid < 512) { data[tid] = sum = sum + data[tid + 512]; }
  __syncthreads();

  if (blockSize >= 512 && tid < 256) { data[tid] = sum = sum + data[tid + 256]; }
  __syncthreads();

  if (blockSize >= 256 && tid < 128) { data[tid] = sum = sum + data[tid + 128]; }
  __syncthreads();

  if (blockSize >= 128 && tid < 64) { data[tid] = sum = sum + data[tid + 64]; }
  __syncthreads();

  if (blockSize >= 64 && tid < 32) { data[tid] = sum = sum + data[tid + 32]; }
  __syncthreads();

  // WarpReduce
  if (tid < 32) { sum = warpReduce<WARP_SIZE>(sum); }

  // block 负责数据的求和结果为 sum，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = sum; }
}

// version7: 在最开始写入 shared memory 之前，进行一次 warp reduction
template<int blockSize>
__global__ void reduce7(float* d_A, const int N) {
  // 申请 shared memory，用于存放 block 负责的数据
  extern __shared__ float data[];

  // 计算索引
  int tid = threadIdx.x;
  int i = blockSize * blockIdx.x + threadIdx.x;

  // 网格跨步循环，对多个元素求和
  float sum = 0.f;
  for (int index = i; index < N; index += blockSize * gridDim.x) { sum += d_A[index]; }

  // 每个 warp 都执行 WarpReduce
  sum = warpReduce<WARP_SIZE>(sum);
  // WarpReduce 结果按照 warp ID 写入 shared memory
  if ((tid & (WARP_SIZE - 1)) == 0) { data[tid / WARP_SIZE] = sum; }

  // 同步，保证所有 WarpReduce 执行完成
  __syncthreads();

  // 只在 warp 0 执行 WarpReduce
  constexpr int NUM_WARPS = CEIL(blockSize, WARP_SIZE);
  if (tid < 32) {
    // 只保留 NUM_WARPS 个有效数据
    sum = tid < NUM_WARPS ? data[tid] : 0.f;
    sum = warpReduce<NUM_WARPS>(sum);
  }

  // block 负责数据的求和结果为 sum，由 0 号 thread 写入 d_A[blockIdx.x] 中
  if (tid == 0) { d_A[blockIdx.x] = sum; }
}

// GPU 求和
void call_reduction_sum_device(int whichKernel, float* d_A, const int N) {
  void (*kernel)(float*, const int);
  const char* kernelName = "";
  const int block = 256;

  switch (whichKernel) {
    case 0:
      kernel = reduce0;
      kernelName = "reduce0";
      break;
    case -1:
      kernel = reduce0_5;
      kernelName = "reduce0_5";
      break;
    case 1:
      kernel = reduce1;
      kernelName = "reduce1";
      break;
    case 2:
      kernel = reduce2;
      kernelName = "reduce2";
      break;
    case 3:
      kernel = reduce3;
      kernelName = "reduce3";
      break;
    case 4:
      kernel = reduce4;
      kernelName = "reduce4";
      break;
    case 5:
      kernel = reduce5<block>;
      kernelName = "reduce5";
      break;
    case 6:
      kernel = reduce6<block>;
      kernelName = "reduce6";
      break;
    case -2:
      kernel = reduce6_vec4<block>;
      kernelName = "reduce6_vec4";
      break;
    case 7:
      kernel = reduce7<block>;
      kernelName = "reduce7";
      break;
    default: break;
  }

  int sharedBytes = block * sizeof(float);
  if (whichKernel == 7) { sharedBytes = CEIL(block, 32) * sizeof(float); }
  int size = N;

  auto getGrid = [](int whichKernel, int size, int block) -> int {
    if (whichKernel == -2) {
      return CEIL(size, 4 * block);
    } else if (whichKernel < 3) {
      return CEIL(size, block);
    } else if (whichKernel < 6) {
      return CEIL(size, block * 2);
    } else {
      return CEIL(size, block * 4);
    }
  };

  while (size > 1) {
    int grid = getGrid(whichKernel, size, block);
    kernel<<<grid, block, sharedBytes>>>(d_A, size);
    printf("kernel: [%s], size: [%d], grid: [%d], block: [%d]\n", kernelName, size, grid, block);

    size = grid;
  }
}

// CPU 求和
float pairwise_sum(float* A, int start, int stop) {
  if (stop == start) { return A[start]; }

  int mid = (start + stop) / 2;
  return pairwise_sum(A, start, mid) + pairwise_sum(A, mid + 1, stop);
}

float call_reduction_sum_host(float* A, const int N) { return pairwise_sum(A, 0, N - 1); }