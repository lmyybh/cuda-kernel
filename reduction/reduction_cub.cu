#include <hipcub/hipcub.hpp>
#include "cmdline.h"
#include "helper_cuda.cuh"
#include "helper_data.h"
#include "reduction.h"

int main(int argc, char* argv[]) {
  cmdline::parser args;
  args.add<int>("N", 'n', "number of elements", false, 16 * 1024 * 1024);
  args.add<unsigned int>("seed", 's', "random seed", false, 0);

  const int N = args.get<int>("N");
  const int nBytes = N * sizeof(float);

  // 初始化数据
  float* A = nullptr;
  checkCudaErrors(hipHostMalloc((void**)&A, nBytes));
  normalInitialData(A, N, args.get<unsigned int>("seed"));

  // CPU 求和
  float sum = call_reduction_sum_host(A, N);
  printf("CPU Sum: %f\n", sum);

  // host -> device
  float* d_A = nullptr;
  checkCudaErrors(hipMalloc((void**)&d_A, nBytes));
  checkCudaErrors(hipMemcpy(d_A, A, nBytes, hipMemcpyHostToDevice));

  // 输出
  float* d_out;
  checkCudaErrors(hipMalloc(&d_out, sizeof(float)));

  // 确定临时存储空间大小
  void* d_temp = nullptr;
  size_t temp_bytes = 0;
  checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp, temp_bytes, d_A, d_out, N));
  checkCudaErrors(hipMalloc(&d_temp, temp_bytes));

  // 分配临时存储空间
  checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp, temp_bytes, d_A, d_out, N));

  // 执行规约求和
  float result;
  checkCudaErrors(hipMemcpy(&result, d_out, sizeof(float), hipMemcpyDeviceToHost));
  printf("Sum: %f\n", result);

  // 释放资源
  checkCudaErrors(hipHostFree(A));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_out));
  checkCudaErrors(hipFree(d_temp));

  return 0;
}